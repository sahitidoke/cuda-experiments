
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__ 
void add(int n, float *x, float *y)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = tid; i < n; i += stride)
    y[i] = x[i] + y[i];
}
int main(void)
{
  int N = 1 << 20;
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  int tpb = 256;
  int bpg = (N + tpb - 1) / tpb;
  add<<<bpg, tpb>>>(N, x, y);
  hipDeviceSynchronize();
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;
  hipFree(x);
  hipFree(y);
  return 0;
}
